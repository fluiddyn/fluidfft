#include "hip/hip_runtime.h"


#include <iostream>
using namespace std;

#include <stdlib.h>

#include <sys/time.h>
#include <fft3d_with_cufft.h>


//  KERNEL CUDA
// Complex scale
static __device__ __host__ inline dcomplex ComplexScale(dcomplex a, myreal s)
{
  dcomplex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

__global__ void vectorNorm(const myreal norm, dcomplex *A, int numElements)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements)
  {
    A[i] = ComplexScale(A[i], norm);
  }
}

////////////////// FIN KERNEL CUDA

FFT3DWithCUFFT::FFT3DWithCUFFT(int argN0, int argN1, int argN2):
  BaseFFT3D::BaseFFT3D(argN0, argN1, argN2)
{
  struct timeval start_time, end_time;
  myreal total_usecs;
  
  this->_init();

  /* y corresponds to dim 0 in physical space */
  /* y corresponds to dim 1 in physical space */
  /* x corresponds to dim 2 in physical space */
  nz = N0;
  ny = N1;
  nx = N2;

  nX0 = N0;
  nX0loc = nX0;
  nX1 = N1;
  nX1loc = nX1;
  nX2 = N2;
  nX2loc = N2;

  nKx = nx/2+1;
  nKy = ny;
  nKz = nz;

  /* This 3D fft is NOT transposed */
  nK0 = nKz;
  nK0loc = nK0;
  nK1 = nKy;
  nK1loc = nK1;
  nK2 = nKx;
  nK2loc = nK2;

  mem_sizer = sizeof(myreal) * N0 * N1 * N2 ;//taille de arrayX
  int new_size = nK0 * nK1 * nK2 ;
  mem_size = 2 * sizeof(myreal) * new_size ;//taille de arrayK

  gettimeofday(&start_time, NULL);
  // Allocate device memory for signal
  checkCudaErrors(hipMalloc((void **)&data, mem_size));
  checkCudaErrors(hipMalloc((void **)&datar, mem_sizer));

  // CUFFT plan
#ifdef SINGLE_PREC
  checkCudaErrors(hipfftPlan3d(&plan, nX0, nX1, nX2, HIPFFT_R2C));
  checkCudaErrors(hipfftPlan3d(&plan1, nX0, nX1, nX2, HIPFFT_C2R));
#else
  checkCudaErrors(hipfftPlan3d(&plan, nX0, nX1, nX2, HIPFFT_D2Z));
  checkCudaErrors(hipfftPlan3d(&plan1, nX0, nX1, nX2, HIPFFT_Z2D));
#endif

  gettimeofday(&end_time, NULL);

  total_usecs = (end_time.tv_sec-start_time.tv_sec) +
    (end_time.tv_usec-start_time.tv_usec)/1000000.;

  if (rank == 0)
    printf("Initialization (%s) done in %f s\n",
        this->get_classname(), total_usecs);
}


void FFT3DWithCUFFT::destroy(void)
{
  // cout << "Object is being destroyed" << endl;
hipFree(data);
hipFree(datar);
hipfftDestroy(plan);
hipfftDestroy(plan1);
}


FFT3DWithCUFFT::~FFT3DWithCUFFT(void)
{
}


char const* FFT3DWithCUFFT::get_classname()
{ return "FFT3DWithCUFFT";}


myreal FFT3DWithCUFFT::compute_energy_from_X(myreal* fieldX)
{
  int ii,jj,kk;
  myreal energy = 0.;
  myreal energy1, energy2;

  for (ii=0; ii<nX0; ii++)
    {
    energy1=0.;
    for (jj=0; jj<nX1; jj++)
      {
      energy2=0.;
      for (kk=0; kk<nX2; kk++)      
        energy2 += pow(fieldX[(ii*nX1+jj)*nX2+kk], 2);
      energy1 += energy2/nX2;
      }
    energy += energy1 / nX1;
    }
  //cout << "energyX=" << energy / nX0 / 2 << endl;

  return (myreal) (energy / nX0 / 2);
}


myreal FFT3DWithCUFFT::compute_energy_from_K(mycomplex* fieldK)
{
  int i0, i1, i2;
  double energy = 0;
  double energy0 = 0;

  // modes i1_seq = iKx = last = nK1 - 1
  i2 = nK2 - 1;
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      energy += (double) square_abs(fieldK[i2 + (i1 + i0*nK1)*nK2]);
      // we must divide by 2 ==> after
  
    energy *= 0.5; //divide by 2!!!

  // other modes
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      for (i2=1; i2<nK2-1; i2++)
        energy += (double) square_abs(fieldK[i2 + (i1 + i0*nK1)*nK2]);
    
  // modes i1_seq = iKx = 0
  i2 = 0;
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      energy0 += (double) square_abs(fieldK[(i1 + i0*nK1)*nK2]);
      // we must divide by 2 ==> after

  energy += energy0/2.;

  //cout << "energyK=" << energy<<  endl;
  return (myreal) energy;
}


myreal FFT3DWithCUFFT::sum_wavenumbers_double(myreal* fieldK)
{
  int i0, i1, i2;
  double sum = 0;
  double sum0 = 0;

  // modes i1_seq = iKx = last = nK1 - 1
  i2 = nK2 - 1;
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      sum += (double) fieldK[i2 + (i1 + i0*nK1)*nK2];
      // we must divide by 2 ==> after
  
    sum *= 0.5; //divide by 2!!!

  // other modes
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      for (i2=1; i2<nK2-1; i2++)
        sum += (double) fieldK[i2 + (i1 + i0*nK1)*nK2];
    
  // modes i1_seq = iKx = 0
  i2 = 0;
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      sum0 += (double) fieldK[(i1 + i0*nK1)*nK2];
      // we must divide by 2 ==> after

  sum += sum0/2.;

  return (myreal) 2.*sum;
}


void FFT3DWithCUFFT::sum_wavenumbers_complex(mycomplex* fieldK, mycomplex* result)
{
  int i0, i1, i2;
  mycomplex sum = 0;
  mycomplex sum0 = 0;

  // modes i1_seq = iKx = last = nK1 - 1
  i2 = nK2 - 1;
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      sum += fieldK[i2 + (i1 + i0*nK1)*nK2];
      // we must divide by 2 ==> after
  
    sum *= 0.5; //divide by 2!!!

  // other modes
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      for (i2=1; i2<nK2-1; i2++)
        sum += fieldK[i2 + (i1 + i0*nK1)*nK2];
    
  // modes i1_seq = iKx = 0
  i2 = 0;
  for (i0=0; i0<nK0; i0++)
    for (i1=0; i1<nK1; i1++)
      sum0 += fieldK[(i1 + i0*nK1)*nK2];
      // we must divide by 2 ==> after

  sum += sum0/2.;

  *result = 2.*sum;
}


myreal FFT3DWithCUFFT::compute_mean_from_K(mycomplex* fieldK)
{
  myreal mean;
  mean = real(fieldK[0]);

  return mean;
}


void FFT3DWithCUFFT::fft(myreal *fieldX, mycomplex *fieldK)
{
  
  
  // cout << "FFT3DWithCUFFT::fft" << endl;
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(datar, fieldX, mem_sizer, hipMemcpyHostToDevice));

  
  // Transform signal and kernel
  //printf("Transforming signal hipfftExecD2Z\n");
#ifdef SINGLE_PREC
  checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)datar, (hipfftComplex *)data));
#else
  checkCudaErrors(hipfftExecD2Z(plan, (hipfftDoubleReal *)datar, (hipfftDoubleComplex *)data));
#endif

  
  // Launch the Vector Norm CUDA Kernel
  myreal norm = inv_coef_norm;
  //  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  int threadsPerBlock = 256;
  int blocksPerGrid =(nK0 * nK1 * nK2 + threadsPerBlock - 1) / threadsPerBlock;
  vectorNorm<<<blocksPerGrid, threadsPerBlock>>>(norm, data, nK0 * nK1 * nK2 );
  

  // Copy host device to memory
  checkCudaErrors(hipMemcpy(fieldK, data, mem_size, hipMemcpyDeviceToHost));


}


void FFT3DWithCUFFT::ifft(mycomplex *fieldK, myreal *fieldX)
{
  //cout << "FFT3DWithCUFFT::ifft" << endl;
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(data, fieldK, mem_size, hipMemcpyHostToDevice));

  // FFT on DEVICE
#ifdef SINGLE_PREC
  checkCudaErrors(hipfftExecC2R(plan1, (hipfftComplex *)data, (hipfftReal *)datar));
#else
  checkCudaErrors(hipfftExecZ2D(plan1, (hipfftDoubleComplex *)data, (hipfftDoubleReal *)datar));
#endif

  // Copy host device to memory
  checkCudaErrors(hipMemcpy(fieldX, datar, mem_sizer, hipMemcpyDeviceToHost));
}

void FFT3DWithCUFFT::ifft_destroy(mycomplex *fieldK, myreal *fieldX)
{
  //cout << "FFT3DWithCUFFT::ifft" << endl;
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(data, fieldK, mem_size, hipMemcpyHostToDevice));

  // FFT on DEVICE
#ifdef SINGLE_PREC
  checkCudaErrors(hipfftExecC2R(plan1, (hipfftComplex *)data, (hipfftReal *)datar));
#else
  checkCudaErrors(hipfftExecZ2D(plan1, (hipfftDoubleComplex *)data, (hipfftDoubleReal *)datar));
#endif

  // Copy host device to memory
  checkCudaErrors(hipMemcpy(fieldX, datar, mem_sizer, hipMemcpyDeviceToHost));
}


