#include "hip/hip_runtime.h"


#include <iostream>
using namespace std;

#include <stdlib.h>

#include <sys/time.h>
#include <fft2d_with_cufft.h>



//  KERNEL CUDA
// Complex scale
static __device__ __host__ inline dcomplex ComplexScale(dcomplex a, real_cu s)
{
  dcomplex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

__global__ void vectorNorm(const real_cu norm, dcomplex *A, int numElements)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements)
  {
    A[i] = ComplexScale(A[i], norm);
  }
}

////////////////// FIN KERNEL CUDA

FFT2DWithCUFFT::FFT2DWithCUFFT(int argN0, int argN1):
  BaseFFT2D::BaseFFT2D(argN0, argN1)
{
  struct timeval start_time, end_time;
  real_cu total_usecs;
  
  this->_init();

 /* y corresponds to dim 0 in physical space */
  /* x corresponds to dim 1 in physical space */
  ny = N0;
  nx = N1;

  nX0 = N0;
  nX0loc = nX0;
  nX1 = N1;
  nX1loc = nX1;
  

  nKx = nx/2+1;
  nKxloc = nKx;
  nKy = ny;
  
  /* This 2D fft is NOT transposed */
  nK0 = nKy;
  nK0loc = nK0;
  nK1 = nKx;
  nK1loc = nK1;
  
  coef_norm = N0*N1;


  mem_sizer = sizeof(real_cu) * N0 * N1 ;//taille de arrayX
  int new_size = nK0 * nK1 ;
  mem_size = 2 * sizeof(real_cu) * new_size ;//taille de arrayK

  gettimeofday(&start_time, NULL);
  // Allocate device memory for signal
  checkCudaErrors(hipMalloc((void **)&data, mem_size));
  checkCudaErrors(hipMalloc((void **)&datar, mem_sizer));

  // CUFFT plan
#ifdef SINGLE_PREC
  checkCudaErrors(hipfftPlan2d(&plan, nX0, nX1, HIPFFT_R2C));
  checkCudaErrors(hipfftPlan2d(&plan1, nX0, nX1, HIPFFT_C2R));
#else
  checkCudaErrors(hipfftPlan2d(&plan, nX0, nX1, HIPFFT_D2Z));
  checkCudaErrors(hipfftPlan2d(&plan1, nX0, nX1, HIPFFT_Z2D));
#endif

  gettimeofday(&end_time, NULL);

  total_usecs = (end_time.tv_sec-start_time.tv_sec) +
    (end_time.tv_usec-start_time.tv_usec)/1000000.;

  if (rank == 0)
    printf("Initialization (%s) done in %f s\n",
        this->get_classname(), total_usecs);
}


void FFT2DWithCUFFT::destroy(void)
{
  // cout << "Object is being destroyed" << endl;
hipFree(data);
hipFree(datar);
hipfftDestroy(plan);
hipfftDestroy(plan1);
}


FFT2DWithCUFFT::~FFT2DWithCUFFT(void)
{
}


char const* FFT2DWithCUFFT::get_classname()
{ return "FFT2DWithCUFFT";}


real_cu FFT2DWithCUFFT::compute_energy_from_X(real_cu* fieldX)
{
  int ii,jj;
  real_cu energy = 0.;
  real_cu energy1;

  for (ii=0; ii<nX0; ii++)
    {
    energy1=0.;
    for (jj=0; jj<nX1; jj++)
      {
      energy1 += pow(fieldX[ii*nX1+jj], 2);
      }
    energy += energy1 / nX1;
    }
  //cout << "energyX=" << energy / nX0 / 2 << endl;

  return energy / nX0 / 2;
}


#ifdef SINGLE_PREC
real_cu FFT2DWithCUFFT::compute_energy_from_K(fftwf_complex* fieldK)
#else
real_cu FFT2DWithCUFFT::compute_energy_from_K(fftw_complex* fieldK)
#endif
{
  int i0, i1;
  double energy = 0;
  double energy0 = 0;

  // modes i1_seq = iKx = last = nK1 - 1
  i1 = nK1 - 1;
  for (i0=0; i0<nK0; i0++)
    energy += (double) pow(cabs(fieldK[i1 + i0*nK1]), 2);//we must divide by 2 ==> after

    energy *= 0.5;//divide by 2!!!

  // other modes
  for (i0=0; i0<nK0; i0++)
    for (i1=1; i1<nK1-1; i1++)
        energy += (double) pow(cabs(fieldK[i1 + i0*nK1]), 2);
    
  // modes i1_seq = iKx = 0
  i1 = 0;
  for (i0=0; i0<nK0; i0++)
    energy0 += (double) pow(cabs(fieldK[i0*nK1]), 2);//we must divide by 2 ==> after

  energy += energy0*0.5;

  //cout << "energyK=" << energy<<  endl;
  return (real_cu) energy;
}


real_cu FFT2DWithCUFFT::compute_mean_from_X(real_cu* fieldX)
{
  real_cu mean,mean1;
  int ii,jj;
  mean=0.;

  for (ii=0; ii<nX0; ii++)
    {
    mean1=0.;
    for (jj=0; jj<nX1; jj++)
      {
      mean1 += fieldX[ii*nX1+jj];
      }
    mean += mean1 / nX1;
    }
  return mean / nX0;
}


#ifdef SINGLE_PREC
real_cu FFT2DWithCUFFT::compute_mean_from_K(fftwf_complex* fieldK)
#else
real_cu FFT2DWithCUFFT::compute_mean_from_K(fftw_complex* fieldK)
#endif
{
  real_cu mean;
  mean = creal(fieldK[0]);

  return mean;
}


#ifdef SINGLE_PREC
void FFT2DWithCUFFT::fft(real_cu *fieldX, fftwf_complex *fieldK)
#else
void FFT2DWithCUFFT::fft(real_cu *fieldX, fftw_complex *fieldK)
#endif
{
  
  
  // cout << "FFT2DWithCUFFT::fft" << endl;
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(datar, fieldX, mem_sizer, hipMemcpyHostToDevice));

  
  // Transform signal and kernel
  //printf("Transforming signal hipfftExecD2Z\n");
#ifdef SINGLE_PREC
  checkCudaErrors(hipfftExecR2C(plan, (hipfftReal *)datar, (hipfftComplex *)data));
#else
  checkCudaErrors(hipfftExecD2Z(plan, (hipfftDoubleReal *)datar, (hipfftDoubleComplex *)data));
#endif

  
  // Launch the Vector Norm CUDA Kernel
  real_cu norm = 1./coef_norm;
  //  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  int threadsPerBlock = 256;
  int blocksPerGrid =(nK0 * nK1 + threadsPerBlock - 1) / threadsPerBlock;
  vectorNorm<<<blocksPerGrid, threadsPerBlock>>>(norm, data, nK0 * nK1 );
  

  // Copy host device to memory
  checkCudaErrors(hipMemcpy(fieldK, data, mem_size, hipMemcpyDeviceToHost));


}


#ifdef SINGLE_PREC
void FFT2DWithCUFFT::ifft(fftwf_complex *fieldK, real_cu *fieldX)
#else
void FFT2DWithCUFFT::ifft(fftw_complex *fieldK, real_cu *fieldX)
#endif
{

  //cout << "FFT2DWithCUFFT::ifft" << endl;
  // Copy host memory to device
  checkCudaErrors(hipMemcpy(data, fieldK, mem_size, hipMemcpyHostToDevice));


  // FFT on DEVICE
#ifdef SINGLE_PREC
  checkCudaErrors(hipfftExecC2R(plan1, (hipfftComplex *)data, (hipfftReal *)datar));
#else
  checkCudaErrors(hipfftExecZ2D(plan1, (hipfftDoubleComplex *)data, (hipfftDoubleReal *)datar));
#endif

  
  // Copy host device to memory
  checkCudaErrors(hipMemcpy(fieldX, datar, mem_sizer, hipMemcpyDeviceToHost));

}


void FFT2DWithCUFFT::init_array_X_random(real_cu* &fieldX)
{
  int ii;
  this->alloc_array_X(fieldX);

  for (ii = 0; ii < nX0*nX1; ++ii)
    fieldX[ii] = (real_cu)rand() / RAND_MAX;
}

